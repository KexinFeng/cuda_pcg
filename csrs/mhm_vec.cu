#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/util/complex.h>
#include "cuda_pcg.h"
#include "utils.h"


#define BLOCK_WIDTH 8  // 8x8, limit 1024: 32x32, 512: 24x24, 256: 16x16, 128: 10x10

namespace cuda_pcg {
template<typename scalar_t>
__global__ void mhm_vec_kernel(
    const float* __restrict__ boson,      // [bs, Ltau * Vs * 2] float32 
    const scalar_t* __restrict__ vec,     // [bs, Ltau * Vs] complex64
    scalar_t* __restrict__ out,           // [bs, Ltau * Vs] complex64
    const int64_t Lx,  // typically Lx^2 = 10x10 = 100, up to 24x24 = 576
    const float dtau)
{
    extern __shared__ scalar_t smem[];  // size: [Lx, Lx] * 2
    scalar_t* interm_vec_in = smem;
    scalar_t* interm_vec_out = &smem[Lx*Lx];
    scalar_t* tmp; 

    int64_t Ltau = gridDim.x;
    int64_t bs = gridDim.y;
    int64_t bw = blockDim.x;

    int64_t stride_vs = Lx * Lx;
    int64_t stride_tau_vs = stride_vs * Ltau;

    int64_t tx = threadIdx.x;  
    int64_t ty = threadIdx.y;
    int64_t tau = blockIdx.x;
    int64_t b = blockIdx.y;

    for (int64_t offset_y = 0; offset_y < ceil_div(Lx, bw); offset_y++) {
        for (int64_t offset_x = 0; offset_x < ceil_div(Lx, bw); offset_x++) {
            int64_t global_x = offset_x * bw + tx;
            int64_t global_y = offset_y * bw + ty;
            if (global_x >= Lx || global_y >= Lx) {
                continue;  // Skip out-of-bound threads
            }
            interm_vec_in[global_y * Lx + global_x] = vec[b * stride_tau_vs + tau * stride_vs + global_y * Lx + global_x];
        }
    }
    __syncthreads();


    // boson [Ltau, Ly, Lx, 2]
    // vec [Ltau, Ly, Lx]
    // center [Lx/2, Lx/2]
    int64_t stride_tau_vs_2 = Ltau * Lx * Lx * 2;
    int64_t stride_vs_2 = Lx * Lx * 2;
    int64_t stride_lx_2 = Lx * 2;

    // // fam4
    for (int64_t cntr_offset_y = 0; cntr_offset_y < ceil_div(Lx, bw); cntr_offset_y++) {
        for (int64_t cntr_offset_x = 0; cntr_offset_x < ceil_div(Lx / 2, bw); cntr_offset_x++) {
            int64_t cntr_x = cntr_offset_x * bw + tx;
            int64_t cntr_y = cntr_offset_y * bw + ty;

            int64_t global_y = cntr_y;
            int64_t global_x = cntr_x * 2 + cntr_y % 2;
            if (global_x >= Lx || global_y >= Lx) {
                continue;
            }

            // fam4: y
            int64_t idx_boson = b * stride_tau_vs_2 + tau * stride_vs_2 + mod(global_y - 1, Lx) * stride_lx_2 + global_x * 2 + 1;
            int64_t i_vec = mod(global_y - 1, Lx) * Lx + global_x;
            int64_t j_vec = global_y * Lx + global_x;

            float boson_val = boson[idx_boson];
            hipFloatComplex cosh_dtau = make_hipFloatComplex(coshf(dtau/2), 0.0f);
            hipFloatComplex sinh_dtau = make_hipFloatComplex(sinhf(dtau/2), 0.0f);
            float cos_boson = cosf(boson_val);
            float sin_boson = sinf(boson_val);
            hipFloatComplex sinh_exp_pos = sinh_dtau * make_hipFloatComplex(cos_boson, sin_boson);  // exp(1i * boson_val)
            hipFloatComplex sinh_exp_neg = sinh_dtau * make_hipFloatComplex(cos_boson, -sin_boson);  // exp(-1i * boson_val)
            if (i_vec < stride_vs && j_vec < stride_vs) {
                interm_vec_out[i_vec] = cosh_dtau * interm_vec_in[i_vec] + sinh_exp_pos * interm_vec_in[j_vec];
                interm_vec_out[j_vec] = cosh_dtau * interm_vec_in[j_vec] + sinh_exp_neg * interm_vec_in[i_vec];
            }
        }
    }
    __syncthreads();

    // // fam3
    tmp = interm_vec_in;
    interm_vec_in = interm_vec_out;
    interm_vec_out = tmp;
    for (int64_t cntr_offset_y = 0; cntr_offset_y < ceil_div(Lx, bw); cntr_offset_y++) {
        for (int64_t cntr_offset_x = 0; cntr_offset_x < ceil_div(Lx / 2, bw); cntr_offset_x++) {
            int64_t cntr_x = cntr_offset_x * bw + tx;
            int64_t cntr_y = cntr_offset_y * bw + ty;

            int64_t global_y = cntr_y;
            int64_t global_x = cntr_x * 2 + cntr_y % 2;
            if (global_x >= Lx || global_y >= Lx) {
                continue;
            }

            // fam3: x
            int64_t idx_boson = b * stride_tau_vs_2 + tau * stride_vs_2 + global_y * stride_lx_2 + mod(global_x - 1, Lx) * 2 + 0;
            int64_t i_vec = global_y * Lx + mod(global_x - 1, Lx);
            int64_t j_vec = global_y * Lx + global_x;

            float boson_val = boson[idx_boson];
            hipFloatComplex cosh_dtau = make_hipFloatComplex(coshf(dtau/2), 0.0f);
            hipFloatComplex sinh_dtau = make_hipFloatComplex(sinhf(dtau/2), 0.0f);
            float cos_boson = cosf(boson_val);
            float sin_boson = sinf(boson_val);
            hipFloatComplex sinh_exp_pos = sinh_dtau * make_hipFloatComplex(cos_boson, sin_boson);  // exp(1i * boson_val)
            hipFloatComplex sinh_exp_neg = sinh_dtau * make_hipFloatComplex(cos_boson, -sin_boson);  // exp(-1i * boson_val)
            if (i_vec < stride_vs && j_vec < stride_vs) {
                interm_vec_out[i_vec] = cosh_dtau * interm_vec_in[i_vec] + sinh_exp_pos * interm_vec_in[j_vec];
                interm_vec_out[j_vec] = cosh_dtau * interm_vec_in[j_vec] + sinh_exp_neg * interm_vec_in[i_vec];
            }
        }
    }
    __syncthreads();

    // fam2
    tmp = interm_vec_in;
    interm_vec_in = interm_vec_out;
    interm_vec_out = tmp;
    for (int64_t cntr_offset_y = 0; cntr_offset_y < ceil_div(Lx, bw); cntr_offset_y++) {
        for (int64_t cntr_offset_x = 0; cntr_offset_x < ceil_div(Lx / 2, bw); cntr_offset_x++) {
            int64_t cntr_x = cntr_offset_x * bw + tx;
            int64_t cntr_y = cntr_offset_y * bw + ty;

            int64_t global_y = cntr_y;
            int64_t global_x = cntr_x * 2 + cntr_y % 2;
            if (global_x >= Lx || global_y >= Lx) {
                continue;
            }

            // fam1: y
            int64_t idx_boson = b * stride_tau_vs_2 + tau * stride_vs_2 + global_y * stride_lx_2 + global_x * 2 + 1;
            int64_t i_vec = global_y * Lx + global_x;
            int64_t j_vec = mod(global_y + 1, Lx) * Lx + global_x;

            float boson_val = boson[idx_boson];
            hipFloatComplex cosh_dtau = make_hipFloatComplex(coshf(dtau/2), 0.0f);
            hipFloatComplex sinh_dtau = make_hipFloatComplex(sinhf(dtau/2), 0.0f);
            float cos_boson = cosf(boson_val);
            float sin_boson = sinf(boson_val);
            hipFloatComplex sinh_exp_pos = sinh_dtau * make_hipFloatComplex(cos_boson, sin_boson);  // exp(1i * boson_val)
            hipFloatComplex sinh_exp_neg = sinh_dtau * make_hipFloatComplex(cos_boson, -sin_boson);  // exp(-1i * boson_val)
            if (i_vec < stride_vs && j_vec < stride_vs) {
                interm_vec_out[i_vec] = cosh_dtau * interm_vec_in[i_vec] + sinh_exp_pos * interm_vec_in[j_vec];
                interm_vec_out[j_vec] = cosh_dtau * interm_vec_in[j_vec] + sinh_exp_neg * interm_vec_in[i_vec];
            }
        }
    }
    __syncthreads();

    // // fam1
    tmp = interm_vec_in;
    interm_vec_in = interm_vec_out;
    interm_vec_out = tmp;
    for (int64_t cntr_offset_y = 0; cntr_offset_y < ceil_div(Lx, bw); cntr_offset_y++) {
        for (int64_t cntr_offset_x = 0; cntr_offset_x < ceil_div(Lx/2, bw); cntr_offset_x++) {
            // Slide the block over the family centers of a rectangle shape [Lx/2, Lx]
            int64_t cntr_x = cntr_offset_x * bw + tx;
            int64_t cntr_y = cntr_offset_y * bw + ty;

            int64_t global_y = cntr_y;
            int64_t global_x = cntr_x * 2 + cntr_y % 2;
            if (global_x >= Lx || global_y >= Lx) {
                continue;  // Skip out-of-bound threads
            }         

            // fam1: x
            int64_t idx_boson = b * stride_tau_vs_2 + tau * stride_vs_2 + global_y * stride_lx_2 + global_x * 2 + 0;
            int64_t i_vec = global_y * Lx + global_x;
            int64_t j_vec = global_y * Lx + mod(global_x + 1, Lx);

            // interm_vec_out[i_vec] = cosh(dtau) * interm_vec_in[i_vec] + sinh(dtau) * exp(1i * boson[idx_boson]) * interm_vec_in[j_vec];
            // interm_vec_out[j_vec] = cosh(dtau) * interm_vec_in[j_vec] + sinh(dtau) * exp(-1i * boson[idx_boson]) * interm_vec_in[i_vec];                    
            float boson_val = boson[idx_boson];
            hipFloatComplex cosh_dtau = make_hipFloatComplex(coshf(dtau), 0.0f);
            hipFloatComplex sinh_dtau = make_hipFloatComplex(sinhf(dtau), 0.0f);
            float cos_boson = cosf(boson_val);
            float sin_boson = sinf(boson_val);
            hipFloatComplex sinh_exp_pos = sinh_dtau * make_hipFloatComplex(cos_boson, sin_boson);  // exp(1i * boson_val)
            hipFloatComplex sinh_exp_neg = sinh_dtau * make_hipFloatComplex(cos_boson, -sin_boson);  // exp(-1i * boson_val)
            if (i_vec < stride_vs && j_vec < stride_vs) {
                interm_vec_out[i_vec] = cosh_dtau * interm_vec_in[i_vec] + sinh_exp_pos * interm_vec_in[j_vec];
                interm_vec_out[j_vec] = cosh_dtau * interm_vec_in[j_vec] + sinh_exp_neg * interm_vec_in[i_vec];
            }
        }
    }
    __syncthreads();

    // fam2
    tmp = interm_vec_in;
    interm_vec_in = interm_vec_out;
    interm_vec_out = tmp;
    for (int64_t cntr_offset_y = 0; cntr_offset_y < ceil_div(Lx, bw); cntr_offset_y++) {
        for (int64_t cntr_offset_x = 0; cntr_offset_x < ceil_div(Lx / 2, bw); cntr_offset_x++) {
            int64_t cntr_x = cntr_offset_x * bw + tx;
            int64_t cntr_y = cntr_offset_y * bw + ty;

            int64_t global_y = cntr_y;
            int64_t global_x = cntr_x * 2 + cntr_y % 2;
            if (global_x >= Lx || global_y >= Lx) {
                continue;
            }

            // fam1: y
            int64_t idx_boson = b * stride_tau_vs_2 + tau * stride_vs_2 + global_y * stride_lx_2 + global_x * 2 + 1;
            int64_t i_vec = global_y * Lx + global_x;
            int64_t j_vec = mod(global_y + 1, Lx) * Lx + global_x;

            float boson_val = boson[idx_boson];
            hipFloatComplex cosh_dtau = make_hipFloatComplex(coshf(dtau/2), 0.0f);
            hipFloatComplex sinh_dtau = make_hipFloatComplex(sinhf(dtau/2), 0.0f);
            float cos_boson = cosf(boson_val);
            float sin_boson = sinf(boson_val);
            hipFloatComplex sinh_exp_pos = sinh_dtau * make_hipFloatComplex(cos_boson, sin_boson);  // exp(1i * boson_val)
            hipFloatComplex sinh_exp_neg = sinh_dtau * make_hipFloatComplex(cos_boson, -sin_boson);  // exp(-1i * boson_val)
            if (i_vec < stride_vs && j_vec < stride_vs) {
                interm_vec_out[i_vec] = cosh_dtau * interm_vec_in[i_vec] + sinh_exp_pos * interm_vec_in[j_vec];
                interm_vec_out[j_vec] = cosh_dtau * interm_vec_in[j_vec] + sinh_exp_neg * interm_vec_in[i_vec];
            }
        }
    }
    __syncthreads();

    // fam3
    tmp = interm_vec_in;
    interm_vec_in = interm_vec_out;
    interm_vec_out = tmp;
    for (int64_t cntr_offset_y = 0; cntr_offset_y < ceil_div(Lx, bw); cntr_offset_y++) {
        for (int64_t cntr_offset_x = 0; cntr_offset_x < ceil_div(Lx / 2, bw); cntr_offset_x++) {
            int64_t cntr_x = cntr_offset_x * bw + tx;
            int64_t cntr_y = cntr_offset_y * bw + ty;

            int64_t global_y = cntr_y;
            int64_t global_x = cntr_x * 2 + cntr_y % 2;
            if (global_x >= Lx || global_y >= Lx) {
                continue;
            }

            // fam3: x
            int64_t idx_boson = b * stride_tau_vs_2 + tau * stride_vs_2 + global_y * stride_lx_2 + mod(global_x - 1, Lx) * 2 + 0;
            int64_t i_vec = global_y * Lx + mod(global_x - 1, Lx);
            int64_t j_vec = global_y * Lx + global_x;

            float boson_val = boson[idx_boson];
            hipFloatComplex cosh_dtau = make_hipFloatComplex(coshf(dtau/2), 0.0f);
            hipFloatComplex sinh_dtau = make_hipFloatComplex(sinhf(dtau/2), 0.0f);
            float cos_boson = cosf(boson_val);
            float sin_boson = sinf(boson_val);
            hipFloatComplex sinh_exp_pos = sinh_dtau * make_hipFloatComplex(cos_boson, sin_boson);  // exp(1i * boson_val)
            hipFloatComplex sinh_exp_neg = sinh_dtau * make_hipFloatComplex(cos_boson, -sin_boson);  // exp(-1i * boson_val)
            if (i_vec < stride_vs && j_vec < stride_vs) {
                interm_vec_out[i_vec] = cosh_dtau * interm_vec_in[i_vec] + sinh_exp_pos * interm_vec_in[j_vec];
                interm_vec_out[j_vec] = cosh_dtau * interm_vec_in[j_vec] + sinh_exp_neg * interm_vec_in[i_vec];
            }
        }
    }
    __syncthreads();

    // fam4
    tmp = interm_vec_in;
    interm_vec_in = interm_vec_out;
    interm_vec_out = tmp;
    for (int64_t cntr_offset_y = 0; cntr_offset_y < ceil_div(Lx, bw); cntr_offset_y++) {
        for (int64_t cntr_offset_x = 0; cntr_offset_x < ceil_div(Lx / 2, bw); cntr_offset_x++) {
            int64_t cntr_x = cntr_offset_x * bw + tx;
            int64_t cntr_y = cntr_offset_y * bw + ty;

            int64_t global_y = cntr_y;
            int64_t global_x = cntr_x * 2 + cntr_y % 2;
            if (global_x >= Lx || global_y >= Lx) {
                continue;
            }

            // fam4: y
            int64_t idx_boson = b * stride_tau_vs_2 + tau * stride_vs_2 + mod(global_y - 1, Lx) * stride_lx_2 + global_x * 2 + 1;
            int64_t i_vec = mod(global_y - 1, Lx) * Lx + global_x;
            int64_t j_vec = global_y * Lx + global_x;

            float boson_val = boson[idx_boson];
            hipFloatComplex cosh_dtau = make_hipFloatComplex(coshf(dtau/2), 0.0f);
            hipFloatComplex sinh_dtau = make_hipFloatComplex(sinhf(dtau/2), 0.0f);
            float cos_boson = cosf(boson_val);
            float sin_boson = sinf(boson_val);
            hipFloatComplex sinh_exp_pos = sinh_dtau * make_hipFloatComplex(cos_boson, sin_boson);  // exp(1i * boson_val)
            hipFloatComplex sinh_exp_neg = sinh_dtau * make_hipFloatComplex(cos_boson, -sin_boson);  // exp(-1i * boson_val)
            if (i_vec < stride_vs && j_vec < stride_vs) {
                interm_vec_out[i_vec] = cosh_dtau * interm_vec_in[i_vec] + sinh_exp_pos * interm_vec_in[j_vec];
                interm_vec_out[j_vec] = cosh_dtau * interm_vec_in[j_vec] + sinh_exp_neg * interm_vec_in[i_vec];
            }
        }
    }
    __syncthreads();

    // Export to out
    for (int64_t offset_y = 0; offset_y < ceil_div(Lx, bw); offset_y++) {
        for (int64_t offset_x = 0; offset_x < ceil_div(Lx, bw); offset_x++) {
            int64_t global_x = offset_x * bw + tx;
            int64_t global_y = offset_y * bw + ty;
            if (global_x >= Lx || global_y >= Lx) {
                continue;  // Skip out-of-bound threads
            }
            out[b * stride_tau_vs + tau * stride_vs + global_y * Lx + global_x] = interm_vec_out[global_y * Lx + global_x];
        }
    }

} // mhm_vec_kernel
} // namespace cuda_pcg

torch::Tensor mhm_vec(
    const torch::Tensor& boson,   // [bs, Ltau * Vs * 2] float32
    const torch::Tensor& vec,     // [bs, Ltau * Vs] complex64
    const int64_t Lx,
    const float dtau)
{
    TORCH_CHECK(boson.dim() == 2, "Boson tensor must have 2 dimensions: [bs, Ltau * Vs * 2]");
    TORCH_CHECK(vec.dim() == 2, "Input tensor must have 2 dimensions: [bs, Ltau * Vs]");
    TORCH_CHECK(boson.size(0) == vec.size(0), "Batch size of boson and vec tensors must match");
    TORCH_CHECK(boson.size(1) == vec.size(1) * 2, "Boson tensor's second dimension must be twice the size of vec's second dimension");

    TORCH_CHECK(vec.is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(boson.is_cuda(), "Boson must  CUDA tensor");
    TORCH_CHECK(vec.scalar_type() == at::ScalarType::ComplexFloat, "Input tensor must be of type ComplexFloat");
    TORCH_CHECK(boson.scalar_type() == at::ScalarType::Float, "Boson tensor must be of type Float");
    TORCH_CHECK(boson.is_contiguous(), "Boson tensor must be contiguous");

    auto out = torch::empty_like(vec);
    auto bs = vec.size(0);
    auto Vs = Lx * Lx;
    auto Ltau = vec.size(1) / Vs; 

    using scalar_t = hipFloatComplex;
    if (vec.dtype() == at::ScalarType::ComplexFloat) {
        using scalar_t = hipFloatComplex; 
    } else if (vec.dtype() == at::ScalarType::ComplexDouble) {
        using scalar_t = hipDoubleComplex;
    } else {
        throw std::invalid_argument("Unsupported data type");
    }

    // B_vec_mul
    dim3 block = {BLOCK_WIDTH, BLOCK_WIDTH};
    dim3 grid = {Ltau, bs};
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    cuda_pcg::mhm_vec_kernel<<<grid, block, 2 * Vs * sizeof(scalar_t), stream>>>(
        reinterpret_cast<float*>(boson.data_ptr()),
        reinterpret_cast<scalar_t*>(vec.data_ptr()),
        reinterpret_cast<scalar_t*>(out.data_ptr()),
        Lx, dtau);
        
    hipError_t kernel_err = hipGetLastError();
    if (kernel_err != hipSuccess) {
        std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(kernel_err) << std::endl;
        throw std::runtime_error("CUDA kernel launch failed");
    }

    hipError_t err = hipStreamSynchronize(stream);
    if (err != hipSuccess) {
        std::cerr << "CUDA stream synchronization failed: " << hipGetErrorString(err) << std::endl;
        throw std::runtime_error("CUDA kernel execution failed");
    }

    // M_vec_mul


    return out;      
}